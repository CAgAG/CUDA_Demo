#include "hip/hip_runtime.h"
// 之前的算法是 kernel 内的并行, cuda stream 是在 kernel 外的并行
//   ==> cuda lib: cudnn cublas tensort
// stream: 一系列的指令执行队列
// multi-stream -- asyn -- order -- asyn  多个流 异步进行 

#include <stdio.h>
#include <math.h>

#define N (1024 * 1024)
#define FULL_SIZE (N * 30)  // 数据规模

// a[] + b[] = c[]
__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if( idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0;

        c[idx] = (as + bs)/2;
    }
}


int main()
{
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if( !prop.deviceOverlap )
    {
        printf("Your device do not support speed up from multi-streams \n");
        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t my_streams[3];  // 定义 3 条流

    int *h_a, *h_b, *h_c;
    // 流数据
    int *d_a0, *d_b0, *d_c0;
    int *d_a1, *d_b1, *d_c1;
    int *d_a2, *d_b2, *d_c2;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStreamCreate(&my_streams[0]);
    hipStreamCreate(&my_streams[1]);
    hipStreamCreate(&my_streams[2]);

    hipMalloc((void**) &d_a0, N * sizeof(int));
    hipMalloc((void**) &d_b0, N * sizeof(int));
    hipMalloc((void**) &d_c0, N * sizeof(int));
    hipMalloc((void**) &d_a1, N * sizeof(int));
    hipMalloc((void**) &d_b1, N * sizeof(int));
    hipMalloc((void**) &d_c1, N * sizeof(int));
    hipMalloc((void**) &d_a2, N * sizeof(int));
    hipMalloc((void**) &d_b2, N * sizeof(int));
    hipMalloc((void**) &d_c2, N * sizeof(int));

    // 流(stream) 要求数据一直驻留在主机, 因此 hipHostAlloc 手动分配主机 CPU 数据
    hipHostAlloc((void**) &h_a, FULL_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &h_b, FULL_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**) &h_c, FULL_SIZE * sizeof(int), hipHostMallocDefault);

    // fill data
    for(int i = 0; i<FULL_SIZE; i++)
    {
        h_a[i] = rand() % 1024;
        h_b[i] = rand() % 1024;
    }

    hipEventRecord(start);
    for(int i = 0; i < FULL_SIZE; i += N * 3)
    {
        hipMemcpyAsync(d_a0, h_a+i, N*sizeof(int), hipMemcpyHostToDevice, my_streams[0]);  // 异步传输数据给到 GPU ==> stream
        hipMemcpyAsync(d_a1, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, my_streams[1]);
        hipMemcpyAsync(d_a2, h_a+i+N+N, N*sizeof(int), hipMemcpyHostToDevice, my_streams[2]);
        hipMemcpyAsync(d_b0, h_a+i, N*sizeof(int), hipMemcpyHostToDevice, my_streams[0]);
        hipMemcpyAsync(d_b1, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, my_streams[1]);
        hipMemcpyAsync(d_b2, h_a+i+N+N, N*sizeof(int), hipMemcpyHostToDevice, my_streams[2]);

        kernel<<<N/256, 256, 0, my_streams[0]>>>(d_a0, d_b0, d_c0);  // 在不同的 stream 中执行
        kernel<<<N/256, 256, 0, my_streams[1]>>>(d_a1, d_b1, d_c1);
        kernel<<<N/256, 256, 0, my_streams[2]>>>(d_a2, d_b2, d_c2);

        hipMemcpyAsync(h_c+i, d_c0, N*sizeof(int), hipMemcpyDeviceToHost, my_streams[0]);
        hipMemcpyAsync(h_c+i+N, d_c1, N*sizeof(int), hipMemcpyDeviceToHost, my_streams[1]);
        hipMemcpyAsync(h_c+i+N+N, d_c2, N*sizeof(int), hipMemcpyDeviceToHost, my_streams[2]);
    }

    hipStreamSynchronize(my_streams[0]);
    hipStreamSynchronize(my_streams[1]);
    hipStreamSynchronize(my_streams[2]);

    hipEventRecord(stop, 0);  // 此处的 0 表示: 第 0 号 stream 结束时, 停止计时
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime , start, stop);
    printf("Time: %3.2f ms\n", elapsedTime);

    // hipFree ...

    return 0;
}
