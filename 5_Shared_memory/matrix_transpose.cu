// 合并访存: 一个线程束内的每个线程之间访问的地址需要是连续的, 而不合并访存则是每个线程之间访问的地址是不连续的
// 连续的线程访问连续的数据
//              matrix transpose: 转置存在问题 ==> 读取是连续的，但是写入不连续 ==> 比如 读 b06 b07 b08 (连续) 对应写一列不连续
//                                                     t56 t57 t58  ==> 线程
//               in        b00 b01 b02 | b03 b04 b05 | b06 b07 b08 
//                         b10 b11 b12 | b13 b14 b15 | b16 b17 b18
//                         b20 b21 b22 | b23 b24 b25 | b26 b27 b28
//                         ------------+-------------+------------
//                         b30 b31 b32 | b33 b34 b35 | b36 b37 b38
//                         b40 b41 b41 | b43 b44 b45 | b46 b47 b48
//                         b50 b51 b52 | b53 b54 b55 | b56 b57 b58   block 1, 2 && thread 2,1  ==> 读取t57
//                                                                   
//
//                         
//               out       b00 b10 b20 | b30 b40 b50
//                         b01 b11 b21 | b31 b41 b51
//                         b02 b12 b22 | b32 b42 b52
//                         ------------+------------
//                         b03 b13 b23 | b33 b43 b53
//                         b04 b14 b24 | b34 b44 b54
//                         b05 b15 b25 | b35 b45 b55
//                         ------------+------------
//                         b06 b16 b26 | b36 b46 b56
//                         b07 b17 b27 | b37 b47 b57
//                         b08 b18 b28 | b38 b48 b58                 block 2, 1 && thread 2,1 ==> 写入t57
// shared memory:  此处当作缓存使用, 在shared memory中进行转置
//  t57 read b57 from global memroy to shared memroy
//  t57 read b48 from shared memory
//  t57 write b48 to global memory


#include <hip/hip_runtime.h>
# include <stdio.h>
# include <math.h>

#define BLOCK_SIZE 32
#define M 3000
#define N 1000

// 在该问题中，并不是一定要使用二维的网格和线程块，因为矩阵中的数据排列本质上依然是一维的。
__managed__ int matrix[N][M];  // N行M列
__managed__ int gpu_result[M][N];
__managed__ int cpu_result[M][N];

void cpu_matrix_transpose(int in[N][M], int out[M][N])
{
    for(int y = 0; y < N; y++)
    {
        for(int x = 0; x < M; x++)
        {
            out[x][y] = in[y][x];
        }
    }
}

__global__ void gpu_matrix_transpose(int in[N][M], int out[M][N])
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if( x < M && y < N)
    {
        out[x][y] = in[y][x];
    }
}

__global__ void gpu_shared_matrix_transpose(int in[N][M], int out[M][N])
{
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    // 对于多维数组， x 维度的线程指标 threadIdx.x 是最内层的（变化最快）
    int x = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ int shared_tmp[BLOCK_SIZE+1][BLOCK_SIZE+1];  // shared_tmp[32] warp

    if(x < M && y < N)
    {
        // 连续的线程读取连续的数据
        shared_tmp[threadIdx.y][threadIdx.x] = in[y][x];
    }
    // 保证数据读取完毕
    __syncthreads();

    /* 以 b57 为例
    in : 读取 b57              ==> 对应 block 1, 2 && thread 2,1
    out: 写入的时候对应的 b48   ==> 对应 block 2, 1 && thread 2,1
    也即 block 内部的 thread 位置没变, 只变了 block 的位置
    in:              out:
       b36 b37 b38       b36 b46 b56      
       b46 b47 b48       b37 b47 b57      
       b56 b57 b58       b38 b48 b58       
    */
    int x1 = threadIdx.x + blockDim.y * blockIdx.y;
    int y1 = threadIdx.y + blockDim.x * blockIdx.x;
    if(x1 < N && y1 < M)
    {
        // 在shared memory中进行转置
        out[y1][x1] = shared_tmp[threadIdx.x][threadIdx.y];  // 32 bank
    }

}


int main()
{
    for(int y=0; y<N; y++)
    {
        for(int x=0; x<M; x++)
        {
            matrix[y][x] = rand()%1024;
        }
    }

    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start);
    hipEventSynchronize(start);

    dim3 dimGrid((M + BLOCK_SIZE - 1)/BLOCK_SIZE, (N + BLOCK_SIZE -1)/BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    for(int i = 0; i < 20; i++)
    {
        gpu_shared_matrix_transpose<<<dimGrid,dimBlock>>>(matrix, gpu_result);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    cpu_matrix_transpose(matrix, cpu_result);

    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool errors = false;
    for(int y = 0; y<M; y++)
    {
        for (int x = 0; x < N; x++)
        {
            if(fabs(cpu_result[y][x] - gpu_result[y][x]) > (1.0e-10))
            {
                errors = true;
            }
        }
        
    }

    printf("Result: %s\n", errors?"Error":"Pass");
    printf("CPU time: %.2f\nGPU time: %.2f\n", time_cpu, time_gpu/20.0);

    return 0;
}