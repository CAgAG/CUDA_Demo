/*
用 __global__ 修饰的函数称为核函数，一般由主机调用，在设备中执行。如果使用动态并行，则也可以在核函数中调用自己或其他核函数。

用 __device__ 修饰的函数叫称为备函数，只能被核函数或其他设备函数调用，在设备中执行。

用 __host__ 修饰的函数就是主机端的普通 C++ 函数，在主机中被调用，在主机中执行。对于主机端的函数，该修饰符可省略。
之所以提供这样一个修饰符，是因为有时可以用 __host__ 和 __device__ 同时修饰一个函数，使得该函数既是一个 C++ 中的普通函数，又是一个设备函数。
这样做可以减少冗余代码。编译器将针对主机和设备分别编译该函数。
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 100000000
#define BLOCK_SIZE 256
#define GRID_SIZE 64
#define topk 20

__managed__ int source[N];
__managed__ int gpu_result[topk];
__managed__ int _1_pass_result[topk * GRID_SIZE];

// topK == 20
// source[N]:  1 + 2 + 3 + 4 + ...............N   
// cpu: for loop 
// gpu: 1 + 2 + 3 + 4 + ...............N    0 + 1 + 2 + 3 + 4[20] + 5 + 6 + 7 
// 如果 source[0] > source[4] 那么 source[0]=source[4] 否则 source[4] = source[0]
//
// thread id step 0:  tid1:source[0] > source[4] -> source[0]
//                    tid1:source[1] > source[5] -> source[1]
//                    tid2:source[2] > source[6] -> source[2]
//                    tid4:source[4] > source[7] -> source[3]
//           step 1:  tid0: source[0] > source[2] -> source[0]
//                    tid1: source[1] > source[3] -> source[1]
//           step 2:  tid0: source[0] > source[1] -> source[0]
// 
// 进一步, 如果是一个二维数组, 例如 第二维有 20 个数
// 那么 tid0:source[0][20] > source[4][20] ==> source[0] & source[4]-> source[0][20]
// 
// 实现思路: 对每个位置的数组进行排序，并始终维护排序的结果, 这样在进行对比赋值时就可以很快
//
// thread id: blockDim.x * blockIdx.x + threadIdx.x + step * blockDim.x * GridDim.x
// thread 0: source[0, 8, 16, 24] sum -> shared memory

// 排序算法: 单步插入排序 ==> 降序
__device__ __host__ void insert_value(int *array, int k, int data)
{
    for(int i=0; i<k; i++)
    {
        // 排除重复的值
        if(array[i] == data)
        {
            return;
        }
    }
    if(data < array[k-1])
    {
        return;
    }
    // 插入排序
    for(int i = k-2; i>=0; i--)
    {
        if(data > array[i])
        {
            array[i + 1] = array[i];
        }
        else
        {
            array[i + 1] = data;
            return;
        }
    }
    
    array[0] = data;
}

__global__ void gpu_topk(int *input, int *output, int length, int k)
{
    __shared__ int shared_memory[BLOCK_SIZE * topk];
    int top_array[topk];

    for(int i = 0; i<topk; i++)
    {
        top_array[i] = INT_MIN;
    }

    for(int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < length; idx += gridDim.x * blockDim.x)
    {
        insert_value(top_array, topk, input[idx]);
    }
    for(int i =0; i<topk; i++)
    {
        shared_memory[topk * threadIdx.x + i] = top_array[i];
    }
    __syncthreads();

    for(int i = BLOCK_SIZE/2; i>=1; i/=2)
    {
        if(threadIdx.x < i)
        {
            // 每个位置都有 topk 个元素, shared_memory[0][topk]
            for(int j=0; j<topk; j++)
            {
                insert_value(top_array, topk, shared_memory[topk *(threadIdx.x + i) + j]);
            }
        }
        __syncthreads();
        if(threadIdx.x < i)
        {
            for(int j=0; j<topk; j++)
            {
                shared_memory[topk * threadIdx.x + j] = top_array[j];
            }
        }
        __syncthreads();
    }
    if(blockIdx.x * blockDim.x < length)
    {
        if(threadIdx.x == 0 )
        {
            for(int i =0; i < topk; i++)
            {
                output[topk * blockIdx.x + i] = shared_memory[i];
            }
        }
    }
}

void cpu_topk(int *input, int *output, int length, int k)
{
    for(int i =0; i< length; i++)
    {
        insert_value(output, k, input[i]);
    }
}

int main()
{
    printf("Init source data...........\n");
    for(int i=0; i<N; i++)
    {
        source[i] = rand();
    }

    printf("Complete init source data.....\n");
    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_gpu);
    hipEventCreate(&stop_cpu);

    hipEventRecord(start);
    hipEventSynchronize(start);
    printf("GPU Run **************\n");
    for(int i =0; i<20; i++)
    {
        gpu_topk<<<GRID_SIZE, BLOCK_SIZE>>>(source, _1_pass_result, N, topk);

        gpu_topk<<<1, BLOCK_SIZE>>>(_1_pass_result, gpu_result, topk * GRID_SIZE, topk);

        hipDeviceSynchronize();
    }
    printf("GPU Complete!!!\n");
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    
    int cpu_result[topk] ={0};
    printf("CPU RUN **************\n");
    cpu_topk(source, cpu_result, N, topk);
    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);
    printf("CPU Complete!!!!!");

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool error = false;
    for(int i =0; i<topk; i++)
    {
        printf("CPU top%d: %d; GPU top%d: %d;\n", i+1, cpu_result[i], i+1, gpu_result[i]);
        if(fabs(gpu_result[i] - cpu_result[i]) > 0)
        {
            error = true;
        }
    }
    printf("Result: %s\n", (error?"Error":"Pass"));
    printf("CPU time: %.2f; GPU time: %.2f\n", time_cpu, (time_gpu/20.0));
}
