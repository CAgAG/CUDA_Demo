#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>  // 协作组
using namespace cooperative_groups;

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

const int NUM_REPEATS = 100;
const int N = 100000000;
const int M = sizeof(real) * N;
const int BLOCK_SIZE = 128;
const unsigned FULL_MASK = 0xffffffff;

void timing(const real *d_x, const int method);

int main(void)
{
    real *h_x = (real *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
    }
    real *d_x;
    hipMalloc(&d_x, M);
    hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice);

    printf("\nusing syncwarp:\n");
    timing(d_x, 0);
    printf("\nusing shfl:\n");
    timing(d_x, 1);
    printf("\nusing cooperative group:\n");
    timing(d_x, 2);

    free(h_x);
    hipFree(d_x);
    return 0;
}

void __global__ reduce_syncwarp(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    for (int offset = 16; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncwarp();
        // 核函数中定义的线程块在执行时将被分配到还没有完全占满的 SM 中。
        // 一个线程块不会被分配到不同的 SM 中，而总是在一个 SM 中，但一个 SM 可以有一个或多个线程块。
        // 一个 SM 以 32 个线程为单位产生、管理、调度、执行线程。这样的 32 个线程称为一个线程束。
        // 一个 SM 可以处理一个或多个线程块。一个线程块又可分为若干个线程束。
        // 例如，一个 128 线程的线程块将被分为 4 个线程束，其中每个线程束包含 32 个具有连续线程号的线程。这样的划分对目前所有的 GPU 架构都是成立的。
        // 当所涉及的线程都在一个线程束内时，可以将线程块同步函数 __syncthreads 换成一个更加廉价的线程束同步函数 __syncwarp。
    }

    if (tid == 0)
    {
        atomicAdd(d_y, s_y[0]);
    }
}

void __global__ reduce_shfl(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    real y = s_y[tid];

    for (int offset = 16; offset > 0; offset >>= 1)
    {
        y += __shfl_down_sync(FULL_MASK, y, offset);
    }

    if (tid == 0)
    {
        atomicAdd(d_y, y);
    }
}

void __global__ reduce_cp(const real *d_x, real *d_y, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    extern __shared__ real s_y[];
    s_y[tid] = (n < N) ? d_x[n] : 0.0;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
    {
        if (tid < offset)
        {
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads();
    }

    real y = s_y[tid];

    thread_block_tile<32> g = tiled_partition<32>(this_thread_block());
    for (int i = g.size() >> 1; i > 0; i >>= 1)
    {
        y += g.shfl_down(y, i);
    }

    if (tid == 0)
    {
        atomicAdd(d_y, y);
    }
}

real reduce(const real *d_x, const int method)
{
    const int grid_size = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const int smem = sizeof(real) * BLOCK_SIZE;

    real h_y[1] = {0};
    real *d_y;
    hipMalloc(&d_y, sizeof(real));
    hipMemcpy(d_y, h_y, sizeof(real), hipMemcpyHostToDevice);

    switch (method)
    {
        case 0:
            reduce_syncwarp<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);
            break;
        case 1:
            reduce_shfl<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);
            break;
        case 2:
            reduce_cp<<<grid_size, BLOCK_SIZE, smem>>>(d_x, d_y, N);
            break;
        default:
            printf("Wrong method.\n");
            exit(1);
    }

    hipMemcpy(h_y, d_y, sizeof(real), hipMemcpyDeviceToHost);
    hipFree(d_y);

    return h_y[0];
}

void timing(const real *d_x, const int method)
{
    real sum = 0;
    
    for (int repeat = 0; repeat < NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipEventQuery(start);

        sum = reduce(d_x, method); 

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        printf("Time = %g ms.\n", elapsed_time);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    printf("sum = %f.\n", sum);
}


