// GPU 代码运行计时


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 32
// example: 
// a[][] * b[][] = c[][]
// 
//  a00 a01 a02 a03    b00 b01 b02 b03    c00 c01 c02 c03
//  a10 a11 a12 a13    b10 b11 b12 b13    c10 c11 c12 c13 
//  a20 a21 a22 a23    b20 b21 b22 b23    c20 c21 c22 c23
//  a30 a31 a32 a33    b30 b31 b32 b33    c30 c31 c32 c33
// 
// 实现
// a的行乘b的列 ==> 使用一维向量存储二维
// c21 = a20 * b01 + a21 * b11 + a22 * b21 + a23 * b31   # c21 ==> y=2(第几行), x=1(第几列)
// a00 a01 a02 a03 a10 a11 a12 a13 a20 a21 a22 a23 a30 a31 a32 a33
// 0   1   2   3   4   5   6   7   8   9   10  11  12  13  14  15
// b00 b01 b02 b03 b10 b11 b12 b13 b20 b21 b22 b23 b30 b31 b32 b33
//
// index = y * size + x   # 宽高 size = 4 
// step 0 -> 3: 
//     a_index = y * size + step;  # 取一行
//     b_index = step * size + x;  # 取一列

void cpu_matrix_mult(int *a, int *b, int *c, const int size)
{
    for(int y=0; y<size; ++y)
    {
        for(int x=0; x<size; ++x)
        {
            int tmp = 0;
            for(int step = 0; step < size; ++step)
            {
                tmp += a[y*size + step] * b[step * size + x];
            }
            c[y * size + x] = tmp;
        }
    }
}

__global__ void gpu_matrix_mult(int *a, int *b, int *c, const int size)
{
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int tmp = 0;
    if( x < size && y < size)
    {
        for( int step = 0; step < size; ++step)
        {
            tmp += a[y * size + step] * b[step * size + x];
        }
        c[y * size + x] = tmp;
    }
}


int main()
{
    int matrix_size = 1000;
    int memsize = sizeof(int) * matrix_size * matrix_size;

    // 使用一维向量存储二维
    int *cpu_a, *cpu_b, *cpu_c, *gpu2cpu_c;
    // cpu的初始化
    hipHostMalloc( (void**)&cpu_a, memsize, hipHostMallocDefault);
    hipHostMalloc( (void**)&cpu_b, memsize, hipHostMallocDefault);
    hipHostMalloc( (void**)&cpu_c, memsize, hipHostMallocDefault);
    hipHostMalloc( (void**)&gpu2cpu_c, memsize, hipHostMallocDefault);

    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            cpu_a[y * matrix_size + x] = rand() % 1024;
        }
    }

    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            cpu_b[y * matrix_size + x] = rand() % 1024;
        }
    }

    int *gpu_a, *gpu_b, *gpu_c;
    // gpu的初始化
    hipMalloc((void**) &gpu_a , memsize);
    hipMalloc((void**) &gpu_b , memsize);
    hipMalloc((void**) &gpu_c , memsize);

    hipMemcpy( gpu_a, cpu_a, memsize, hipMemcpyHostToDevice);
    hipMemcpy( gpu_b,cpu_b, memsize, hipMemcpyHostToDevice);

    // cuda event
    hipEvent_t start, stop_cpu, stop_gpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_cpu);
    hipEventCreate(&stop_gpu);

    // 开始计时
    hipEventRecord(start);
    // cudaEventSynchronize(start) 或者使用 cudaEventQuery(start) // 如果前面没有同步函数的话, 就需要添加这个进行同步 ==> cudaMemcpy 就是一个同步函数
    /*
    正常情况下在每一个cudaEventRecord(stop);下面都要接上cudaEventSynchronize(stop);
    cudaEventRecord(stop_gpu);
    cudaEventSynchronize(stop_gpu);
    win vista+之后的系统里面有一个叫wddm的模式, 它会把GPU上的任务延迟一下, 攒多了一起给到GPU, 所以实际上它的执行时间上可能会有问题, 写程序要有严格的同步函数. 
    一般linux的是tcc模式, TCC驱动允许不进行系统调用，不切换进OS的内核态，就能直接从用户态发布命令给显卡，此时不仅仅降低了CPU使用，
    也减轻了kernel启动延迟（因为CPU上不需要进行昂贵的系统调用了）。
    因此在TCC驱动下，有更好的整体性能和kernel启动延迟。这是TCC的一大好处。
    */

    unsigned int grid_rows = (matrix_size + BLOCK_SIZE -1)/BLOCK_SIZE;
    unsigned int grid_cols = (matrix_size + BLOCK_SIZE -1)/BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    // 1. gpu warp = 32
    // 2. BLOCK_SIZE * BLOCK_SIZE <= 1024
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // gpu compute
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c, matrix_size);
    hipMemcpy(gpu2cpu_c, gpu_c, memsize, hipMemcpyDeviceToHost);
    // 结束计时，gpu
    hipEventRecord(stop_gpu);
    // cpu compute
    cpu_matrix_mult(cpu_a, cpu_b, cpu_c, matrix_size);
    // 结束计时，cpu
    hipEventRecord(stop_cpu);

    // 统计时间
    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);
    // GPU time: 2.58 ms
    // CPU time: 6.21 ms
    printf("GPU time: %.2f ms\n", time_gpu);
    printf("CPU time: %.2f ms\n", time_cpu);
    hipEventDestroy(start);
    hipEventDestroy(stop_gpu);
    hipEventDestroy(stop_cpu);

    // evaluate
    bool errors = false;
    for(int y=0; y<matrix_size; ++y)
    {
        for(int x=0; x<matrix_size; ++x)
        {
            if(fabs(gpu2cpu_c[y*matrix_size + x] - cpu_c[y*matrix_size + x]) > (1.0e-10))
            {
                //printf("%d, %d\n", y, x);
                errors = true;
            }
        }
    }
    printf("Result: %s\n", errors?"Errors":"Pass");

    hipHostFree(cpu_a);
    hipHostFree(cpu_b);
    hipHostFree(cpu_c);
    hipHostFree(gpu2cpu_c);
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}

// 在 CUDA 工具箱中有一个称为 nvprof 的可执行文件，可用于对 CUDA 程序进行更多的性能剖析。
// 简易使用: nvprof ./a.out
//    或者:  nvprof --unified-memory-profiling off ./a.out